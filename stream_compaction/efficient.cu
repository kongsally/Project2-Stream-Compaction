#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
namespace Efficient {

#define blockSize 128
int *temp_scan;
int *scan_result;

__global__ void upSweep(int n, int d, int *o_data, int *i_data) {
	int index =  (blockIdx.x * blockDim.x) + threadIdx.x;	
	if (index <= n) {
		if (index % (int)pow(2.0, d+1) == 0) {
			o_data[index-1] = i_data[index - 1 - (int)pow(2.0, d)] + i_data[index - 1];
		} 
	}
}

__global__ void downSweep(int n, int d, int *o_data, int *i_data) {
	int index =  (blockIdx.x * blockDim.x) + threadIdx.x;
	int temp = 0;
	if (index <= n) {
		if (index % (int)pow(2.0, d+1) == 0) {
			temp = i_data[index - 1 - (int)pow(2.0, d)];
			o_data[index - 1 - (int)pow(2.0, d)] = i_data[index-1];
			o_data[index-1] = temp + i_data[index - 1];
		} 
	}

}

void scan(int n, int *odata, const int *idata) {
    int d = ilog2ceil(n);
	int total = (int) pow(2.0, d);

	hipMalloc((void**)&scan_result, total * sizeof(int));
	hipMalloc((void**)&temp_scan, total * sizeof(int));
	
	hipMemcpy(temp_scan, idata, total * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(scan_result, idata, total * sizeof(int), hipMemcpyHostToDevice);

	dim3 fullBlocksPerGrid((total + blockSize - 1) / blockSize);
	
	for (int i = 0; i < d; i++) {
		upSweep<<<fullBlocksPerGrid, blockSize>>>(total, i, scan_result, temp_scan);
		temp_scan = scan_result;
	}

	
	hipMemcpy(odata, scan_result, total * sizeof(int), hipMemcpyDeviceToHost);
	odata[total-1] = 0;

	hipMemcpy(scan_result, odata, total * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(temp_scan, odata, total * sizeof(int), hipMemcpyHostToDevice);

	for (int i = d-1; i >= 0; i--) {
		downSweep<<<fullBlocksPerGrid, blockSize>>>(total, i, scan_result, temp_scan);
		temp_scan = scan_result;
	}

	hipMemcpy(odata, scan_result, total * sizeof(int), hipMemcpyDeviceToHost);
	hipFree(scan_result);
	hipFree(temp_scan);

}

/**
 * Performs stream compaction on idata, storing the result into odata.
 * All zeroes are discarded.
 *
 * @param n      The number of elements in idata.
 * @param odata  The array into which to store elements.
 * @param idata  The array of elements to compact.
 * @returns      The number of elements remaining after compaction.
 */
int compact(int n, int *odata, const int *idata) {
    // TODO
    return -1;
}

}
}
