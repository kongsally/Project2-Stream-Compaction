#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
namespace Efficient {

#define blockSize 128
int *temp_scan;
int *scan_result;

__global__ void upSweep(int n, int d, int *o_data, int *i_data) {
	int index =  (blockIdx.x * blockDim.x) + threadIdx.x;	
	if (index <= n) {
		if (index % (int)pow(2.0, d+1) == 0) {
			o_data[index-1] = i_data[index - 1 - (int)pow(2.0, d)] + i_data[index - 1];
		} 
	}
}

__global__ void downSweep(int n, int d, int *o_data, int *i_data) {
	int index =  (blockIdx.x * blockDim.x) + threadIdx.x;
	int temp = 0;
	if (index <= n) {
		if (index % (int)pow(2.0, d+1) == 0) {
			temp = i_data[index - 1 - (int)pow(2.0, d)];
			o_data[index - 1 - (int)pow(2.0, d)] = i_data[index-1];
			o_data[index-1] = temp + i_data[index - 1];
		} 
	}

}

void scan(int n, int *odata, const int *idata) {
    int d = ilog2ceil(n);

	hipMalloc((void**)&scan_result, n * sizeof(int));
	hipMalloc((void**)&temp_scan, n * sizeof(int));
	
	hipMemcpy(temp_scan, idata, n * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(scan_result, idata, n * sizeof(int), hipMemcpyHostToDevice);

	dim3 fullBlocksPerGrid((n + blockSize - 1) / blockSize);
	
	for (int i = 0; i < d; i++) {
		upSweep<<<fullBlocksPerGrid, blockSize>>>(n, i, scan_result, temp_scan);
		temp_scan = scan_result;
	}

	
	hipMemcpy(odata, scan_result, n * sizeof(int), hipMemcpyDeviceToHost);
	odata[n-1] = 0;

	hipMemcpy(scan_result, odata, n * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(temp_scan, odata, n * sizeof(int), hipMemcpyHostToDevice);

	for (int i = d-1; i >= 0; i--) {
		downSweep<<<fullBlocksPerGrid, blockSize>>>(n, i, scan_result, temp_scan);
		temp_scan = scan_result;
	}

	hipMemcpy(odata, scan_result, n * sizeof(int), hipMemcpyDeviceToHost);
	hipFree(scan_result);
	hipFree(temp_scan);

}

/**
 * Performs stream compaction on idata, storing the result into odata.
 * All zeroes are discarded.
 *
 * @param n      The number of elements in idata.
 * @param odata  The array into which to store elements.
 * @param idata  The array of elements to compact.
 * @returns      The number of elements remaining after compaction.
 */
int compact(int n, int *odata, const int *idata) {
    // TODO
    return -1;
}

}
}
